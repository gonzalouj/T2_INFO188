#include "hip/hip_runtime.h"
// =============================================================================
// INFO188: Tarea 2 - Batalla de sorting paralelo
// Implementación de algoritmos de ordenamiento paralelo para CPU y GPU
// =============================================================================

#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <random>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// Definimos threads por bloque según recomendaciones de NVIDIA para RTX/GTX
#define THREADS_PER_BLOCK 256

// Funciones auxiliares
bool check_sorting(int* arr, long long n) {
    for(long long i = 1; i < n; i++) {
        if(arr[i] < arr[i-1]) return false;
    }
    return true;
}

void generate_random_data(int *arr, long long n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dis(0, n * 2);
    
    for(long long i = 0; i < n; i++) {
        arr[i] = dis(gen);
    }
}

// =================== ALGORITMO 1: ORDENAMIENTO POR MEZCLA PARALELO (CPU) ===================
/*
 * Implementación paralela de Ordenamiento por Mezcla usando OpenMP
 * El algoritmo divide el arreglo en segmentos y los procesa en paralelo
 * usando múltiples hilos de CPU. La fase de mezcla se realiza
 * de manera iterativa, combinando pares de subarreglos ordenados.
 */

/**
 * Función merge: Combina dos subarreglos ordenados en uno solo
 * @param arr: Arreglo original
 * @param temp: Arreglo temporal para la mezcla
 * @param start: Índice inicial
 * @param mid: Índice medio (donde comienza el segundo subarreglo)
 * @param end: Índice final
 */
void merge(int *arr, int *temp, int start, int mid, int end) {
    int i = start, j = mid, k = start;
    
    while (i < mid && j < end) {
        if (arr[i] <= arr[j]) {
            temp[k++] = arr[i++];
        } else {
            temp[k++] = arr[j++];
        }
    }
    
    while (i < mid) {
        temp[k++] = arr[i++];
    }
    while (j < end) {
        temp[k++] = arr[j++];
    }
    
    for (i = start; i < end; i++) {
        arr[i] = temp[i];
    }
}

/**
 * Implementación paralela del Merge Sort
 * - Usa OpenMP para paralelizar la fase de merge
 * - Cada thread procesa un chunk del array independientemente
 * @param arr: Array a ordenar
 * @param temp: Array temporal necesario para el merge
 * @param n: Tamaño del array
 * @param num_threads: Número de threads a usar
 */
void parallel_merge_sort(int *arr, int *temp, int n, int num_threads) {
    omp_set_num_threads(num_threads);
    
    for (int chunk_size = 1; chunk_size < n; chunk_size *= 2) {
        #pragma omp parallel for
        for (int i = 0; i < n; i += 2 * chunk_size) {
            int start = i;
            int mid = std::min(i + chunk_size, n);
            int end = std::min(i + 2 * chunk_size, n);
            merge(arr, temp, start, mid, end);
        }
    }
}

// =================== ALGORITMO 2: ORDENAMIENTO RADIX (GPU) ===================
/*
 * Implementación de Radix Sort en GPU utilizando CUDA Thrust
 * El algoritmo aprovecha la biblioteca Thrust que provee una 
 * implementación optimizada del ordenamiento Radix para GPUs NVIDIA.
 */

/**
 * Wrapper para el Radix Sort de Thrust
 * Maneja la transferencia de memoria entre CPU y GPU, y la ejecución
 * del algoritmo de ordenamiento en la GPU
 * @param data: Array a ordenar
 * @param n: Tamaño del array
 * @param num_blocks: Número de bloques CUDA a usar
 */
void radix_sort_cuda(int* data, int n, int num_blocks) {
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    hipMemcpy(d_data, data, n * sizeof(int), hipMemcpyHostToDevice);
    
    // Usar thrust::sort que implementa Radix Sort paralelo para enteros
    thrust::device_ptr<int> dev_ptr(d_data);
    thrust::sort(dev_ptr, dev_ptr + n);
    
    hipMemcpy(data, d_data, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data);
}

/**
 * Programa principal:
 * Maneja la lógica de entrada, memoria y ejecución de los algoritmos
 * Uso: ./prog <n> <modo> <hilos>
 * - n: tamaño del arreglo
 * - modo: 0 para CPU (Ordenamiento por Mezcla), 1 para GPU (Radix Sort)
 * - hilos: número de hilos para CPU (ignorado en GPU)
 */
int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Uso: %s <n> <modo> <hilos>\n", argv[0]);
        printf("  n: tamaño del arreglo\n");
        printf("  modo: CPU -> 0, GPU -> 1\n");
        printf("  hilos: número de hilos CPU\n");
        return 1;
    }
    
    long long n = atoll(argv[1]);
    int mode = atoi(argv[2]);
    int threads = atoi(argv[3]);
    
    if (n <= 0 || (mode != 0 && mode != 1) || threads <= 0) {
        printf("Error: argumentos inválidos\n");
        return 1;
    }

    int *data = (int *)malloc(n * sizeof(int));
    if (data == NULL) {
        printf("Error: no se pudo asignar memoria\n");
        return 1;
    }

    generate_random_data(data, n);

    if (n <= 20) {
        printf("Arreglo original: ");
        for(int i = 0; i < n; i++) printf("%d ", data[i]);
        printf("\n");
    }

    int *temp = (int *)malloc(n * sizeof(int));
    if (temp == NULL) {
        printf("Error: no se pudo asignar memoria temporal\n");
        free(data);
        return 1;
    }

    double start = omp_get_wtime();
    
    if (mode == 0) {
        parallel_merge_sort(data, temp, n, threads);
    } else {
        int num_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        radix_sort_cuda(data, n, num_blocks);
    }
    
    double end = omp_get_wtime();
    double total_time = end - start;
    
    if (n <= 20) {
        printf("Arreglo ordenado: ");
        for(int i = 0; i < n; i++) printf("%d ", data[i]);
        printf("\n");
    }

    if (!check_sorting(data, n)) {
        printf("Error: el arreglo no está ordenado correctamente\n");
        return 1;
    }
    
    printf("%.4f\n", total_time);

    free(temp);
    free(data);
    return 0;
}